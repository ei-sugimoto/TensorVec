#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm.h"
#include "gemv/gemv.cuh"

#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "utils/cublas_utils.h"

using data_type = double;

int main(int argc, char *argv[])
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 8;
    const int n = 8;
    const int lda = m;

    /*
     *   A = | 1.0 2.0 |
     *       | 3.0 4.0 |
     *   x = | 5.0 6.0 |
     */

    std::vector<data_type> A(m * n, 0); // サイズを m*n に設定し、全要素を0で初期化
    std::vector<data_type> x(n, 0);     // サイズを n に設定し、全要素を0で初期化
    std::vector<data_type> y(m, 0);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;
    const int incx = 1;
    const int incy = 1;

    loop(i, m)
    {
        loop(j, n)
        {
            A[i * m + j] = i * n + j;
        }
    }

    loop(i, n)
    {
        x.at(i) = i + 1;
    }

    data_type *d_A = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;

    printf("A\n");
    print_matrix(m, n, A.data(), lda);
    printf("=====\n");

    printf("x\n");
    print_vector(x.size(), x.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(data_type) * x.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(
        hipblasDgemv(cublasH, transa, m, n, &alpha, d_A, lda, d_x, incx, &beta, d_y, incy));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(data_type) * y.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   y = | 17.00 39.00 |
     */

    printf("y\n");
    print_vector(y.size(), y.data());
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}