#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)
#define space() cout << "===============================================" << endl

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include "../utils/print_matrix.h"
#include "mm_gpu80.cuh"
#include "../utils/timer.hpp"
#include "../utils/operate_matrix.cuh"

int main()
{
    bool isPrint = true;
    const int thresholdMatrixSize = 16;
    const int m = 8;
    class Timer timer;

    typeM *deviceM, *M;
    typeV *deviceV, *deviceRes, *V, *res;

    M = (typeM *)malloc(sizeof(typeM) * m * m);
    V = (typeV *)malloc(sizeof(typeV) * m * m);
    res = (typeV *)malloc(sizeof(typeV) * m * m);

    if (m < thresholdMatrixSize)
    {
        isPrint = true;
    }
    else
    {
        isPrint = false;
    }

    hipMalloc((void **)&deviceM, sizeof(typeM) * m * m);
    hipMalloc((void **)&deviceV, sizeof(typeV) * m * m);
    hipMalloc((void **)&deviceRes, sizeof(typeV) * m * m);

    create_matrix<typeM><<<dim3((m + 16 - 1) / 16, (m + 16 - 1) / 16), dim3(16, 16)>>>(deviceM, m, m);
    create_vector_for_cutlass<typeV><<<dim3((m + 16 - 1) / 16, (m + 16 - 1) / 16), dim3(16, 16)>>>(deviceV, m, m);

    hipMemcpy(M, deviceM, sizeof(typeM) * m * m, hipMemcpyDeviceToHost);
    hipMemcpy(V, deviceV, sizeof(typeV) * m * m, hipMemcpyDeviceToHost);

    if (isPrint)
    {
        print_matrix(m, m, M, m);
        space();
        print_matrix(m, m, V, m);
    }

    timer.reset();
    mm_gpu<typeM, typeV>(m, m, m, 1.0f, deviceM, deviceV, 1.0f, deviceRes);
    timer.stop();
    space();
    timer.print();

    hipMemcpy(res, deviceRes, sizeof(typeV) * m * m, hipMemcpyDeviceToHost);

    if (isPrint)
    {
        space();

        print_matrix(m, m, res, m);
    }
    return 0;
}