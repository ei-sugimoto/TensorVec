#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)
#define space() cout << "===============================================" << endl

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include "../utils/print_matrix.h"
#include "mm_gpu80.cuh"
#include "../utils/timer.hpp"

int main()
{
    bool isPrint = true;
    const int thresholdMatrixSize = 16;
    const int m = 32;
    class Timer timer;

    typeM *deviceM, *M;
    typeV *deviceV, *deviceRes, *V, *res;

    M = (typeM *)malloc(sizeof(typeM) * m * m);
    V = (typeV *)malloc(sizeof(typeV) * m * m);
    res = (typeV *)malloc(sizeof(typeV) * m * m);

    if (m < thresholdMatrixSize)
    {
        isPrint = true;
    }
    else
    {
        isPrint = false;
    }

    loop(i, m)
    {
        loop(j, m)
        {
            M[i * m + j] = i * m + j;
        }
    }
    if (isPrint)
    {
        space();

        print_matrix(m, m, M, m);
    }

    loop(i, m)
    {
        loop(j, m)
        {
            V[i + j * m] = j;
        }
    }

    if (isPrint)
    {
        space();

        print_matrix(m, m, V, m);
    }

    hipMalloc((void **)&deviceM, sizeof(typeM) * m * m);
    hipMalloc((void **)&deviceV, sizeof(typeV) * m * m);
    hipMalloc((void **)&deviceRes, sizeof(typeV) * m * m);

    hipMemcpy(deviceM, M, sizeof(typeM) * m * m, hipMemcpyHostToDevice);
    hipMemcpy(deviceV, V, sizeof(typeV) * m * m, hipMemcpyHostToDevice);
    timer.reset();
    mm_gpu<typeM, typeV>(m, m, m, 1.0f, deviceM, deviceV, 1.0f, deviceRes);
    timer.stop();
    space();
    timer.print();

    hipMemcpy(res, deviceRes, sizeof(typeV) * m * m, hipMemcpyDeviceToHost);

    if (isPrint)
    {
        space();

        print_matrix(m, m, res, m);
    }
    return 0;
}