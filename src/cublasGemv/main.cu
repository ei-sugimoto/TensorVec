#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)
#define space() cout << "===============================================" << endl

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../utils/cublas_utils.h"
#include "../utils/timer.hpp"

using data_type = double;

int main(int argc, char *argv[])
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 32;
    const int n = 32;
    const int lda = m;
    Timer timer;

    std::vector<data_type> A(m * n, 0);
    std::vector<data_type> x(n, 0);
    std::vector<data_type> y(m, 0);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;
    const int incx = 1;
    const int incy = 1;

    loop(i, m)
    {
        loop(j, n)
        {
            A[i * m + j] = i * n + j;
        }
    }

    loop(i, n)
    {
        x.at(i) = i;
    }

    data_type *d_A = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;

    printf("A\n");
    print_matrix(m, n, A.data(), lda);
    space();

    printf("x\n");
    print_vector(x.size(), x.data());
    space();

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(data_type) * x.size(), hipMemcpyHostToDevice,
                               stream));

    timer.reset();
    CUBLAS_CHECK(
        hipblasDgemv(cublasH, transa, m, n, &alpha, d_A, lda, d_x, incx, &beta, d_y, incy));
    timer.stop();
    timer.print();
    space();

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(data_type) * y.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   y = | 17.00 39.00 |
     */

    printf("y\n");
    print_vector(y.size(), y.data());
    space();
    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}