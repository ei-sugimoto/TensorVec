#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)
#define space() cout << "===============================================" << endl

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../utils/cublas_utils.h"
#include "../utils/timer.hpp"
#include "../utils/operate_matrix.cuh"

using data_type = double;

int main(int argc, char *argv[])
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    bool isPrint = true;
    const int thresholdMatrixSize = 16;

    const int m = 4352;
    const int n = 4352;
    const int lda = m;
    Timer timer;

    std::vector<data_type> A(m * n, 0);
    std::vector<data_type> x(n, 0);
    std::vector<data_type> y(m, 0);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;
    const int incx = 1;
    const int incy = 1;

    if (m < thresholdMatrixSize)
    {
        isPrint = true;
    }
    else
    {
        isPrint = false;
    }

    data_type *d_A = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    create_matrix<data_type><<<dim3((m + 16 - 1) / 16, (n + 16 - 1) / 16), dim3(16, 16)>>>(d_A, m, n);
    create_vector<data_type><<<dim3((n + 16 - 1) / 16, (1 + 16 - 1) / 16), dim3(16, 16)>>>(d_x, n);
    CUDA_CHECK(hipMemcpy(A.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(x.data(), d_x, sizeof(data_type) * x.size(), hipMemcpyDeviceToHost));

    if (isPrint)
    {
        space();
        printf("A\n");
        print_matrix(m, n, A.data(), lda);
        space();
    }

    if (isPrint)
    {
        space();
        printf("x\n");
        print_vector(x.size(), x.data());
        space();
    }

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    hipblasOperation_t transa = HIPBLAS_OP_T;

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    timer.reset();
    CUBLAS_CHECK(
        hipblasDgemv(cublasH, transa, m, n, &alpha, d_A, lda, d_x, incx, &beta, d_y, incy));
    hipDeviceSynchronize();

    timer.stop();
    space();
    cout << "m : " << m << endl;
    timer.print();
    space();

    CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(data_type) * y.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (isPrint)
    {
        space();
        printf("y\n");
        print_vector(y.size(), y.data());
        space();
    }
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
