#define ll long long
#define loop(i, n) for (ll i = 0; i < n; i++)
#define space() cout << "===============================================" << endl

typedef float typeM;
typedef float typeV;
using namespace std;

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../utils/cublas_utils.h"
#include "../utils/timer.hpp"

using data_type = double;

int main(int argc, char *argv[])
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    bool isPrint = true;
    const int thresholdMatrixSize = 16;

    const int m = 128;
    const int n = 128;
    const int lda = m;
    Timer timer;

    std::vector<data_type> A(m * n, 0);
    std::vector<data_type> x(n, 0);
    std::vector<data_type> y(m, 0);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;
    const int incx = 1;
    const int incy = 1;

    if (m < thresholdMatrixSize)
    {
        isPrint = true;
    }
    else
    {
        isPrint = false;
    }

    loop(i, m)
    {
        loop(j, n)
        {
            A[i * m + j] = i * n + j;
        }
    }

    loop(i, n)
    {
        x.at(i) = i;
    }

    data_type *d_A = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;

    if (isPrint)
    {
        space();
        printf("A\n");
        print_matrix(m, n, A.data(), lda);
        space();
    }

    if (isPrint)
    {
        space();
        printf("x\n");
        print_vector(x.size(), x.data());
        space();
    }

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(data_type) * x.size(), hipMemcpyHostToDevice,
                               stream));

    timer.reset();
    CUBLAS_CHECK(
        hipblasDgemv(cublasH, transa, m, n, &alpha, d_A, lda, d_x, incx, &beta, d_y, incy));
    timer.stop();
    space();
    timer.print();
    space();

    CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(data_type) * y.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (isPrint)
    {
        space();
        printf("y\n");
        print_vector(y.size(), y.data());
        space();
    }
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
